#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

__global__ void kernel(int *data_persistent, int *data_stream, int dataSize, int freqSize) {
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	data_persistent[tid % freqSize] = 2 * data_persistent[tid % freqSize];
	data_stream[tid % dataSize] = 2 * data_stream[tid % dataSize];
}

int main() {
	// create cuda stream
	hipStream_t stream;
	hipStreamCreate(&stream);

	// cuda device properties variable
	hipDeviceProp_t prop;
	int device_id;
	hipGetDeviceProperties(&prop, device_id);
	size_t size = min(int(prop.l2CacheSize * 0.75), prop.persistingL2CacheMaxSize);
	int freqSize = 1LL << 10;
	int *data_persistent;
	hipMallocManaged(&data_persistent, freqSize * (1LL << 20) * sizeof(int));
	hipLaunchAttributeValue stream_attribute;
	stream_attribute.accessPolicyWindow.base_ptr = reinterpret_cast<void*>(data_persistent);
	stream_attribute.accessPolicyWindow.num_bytes = freqSize * sizeof(int);
	stream_attribute.accessPolicyWindow.hitRatio = 1.0;
	return 0;
}
